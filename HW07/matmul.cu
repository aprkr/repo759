#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

template <typename T>
__host__ void matmul_common(const T *A, const T *B, T *C, unsigned int n, unsigned int block_dim) {
    T *d_A, *d_B, *d_C;
    size_t size = n * n * sizeof(T);

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);

    // Start timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the kernel
    matmul_kernel<T><<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n, block_dim);

    // Stop timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time taken: " << milliseconds << " ms" << std::endl;

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim) {
    matmul_common(A, B, C, n, block_dim);
}

__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n, unsigned int block_dim) {
    matmul_common(A, B, C, n, block_dim);
}

__host__ void matmul_3(const double *A, const double *B, double *C, unsigned int n, unsigned int block_dim) {
    matmul_common(A, B, C, n, block_dim);
}

__global__ void matmul_kernel(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim) {
    // Tile size
    __shared__ int As[block_dim][block_dim];
    __shared__ int Bs[block_dim][block_dim];

    // Global thread indices
    unsigned int row = blockIdx.y * block_dim + threadIdx.y;
    unsigned int col = blockIdx.x * block_dim + threadIdx.x;
    
    int value = 0;

    // Loop over sub-matrices to compute the product
    for (unsigned int m = 0; m < (n + block_dim - 1) / block_dim; m++) {
        // Load the data into shared memory
        if (row < n && (m * block_dim + threadIdx.x) < n)
            As[threadIdx.y][threadIdx.x] = A[row * n + m * block_dim + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0;

        if (col < n && (m * block_dim + threadIdx.y) < n)
            Bs[threadIdx.y][threadIdx.x] = B[(m * block_dim + threadIdx.y) * n + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        // Perform partial multiplication
        for (unsigned int k = 0; k < block_dim; k++) {
            value += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Store the result in C
    if (row < n && col < n)
        C[row * n + col] = value;
}
