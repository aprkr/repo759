
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <random>

// CUDA kernel that calculates ax + y
__global__ void computeValues(int *dA, int a) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;  // Unique thread index
    if (index < 16) {
        int x = threadIdx.x;
        int y = blockIdx.x;
        dA[index] = a * x + y;
    }
}

int main() {
    int some_seed = 759;
    std::mt19937 generator(some_seed);
    int a = generator() % 10;
    
    printf("Random integer a = %d\n", a);

    int hA[16];
    
    int *dA;
    hipMalloc((void**)&dA, 16 * sizeof(int));

    computeValues<<<2, 8>>>(dA, a);

    hipMemcpy(hA, dA, 16 * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    printf("Values in hA: ");
    for (int i = 0; i < 16; ++i) {
        printf("%d ", hA[i]);
    }
    printf("\n");

    hipFree(dA);

    return 0;
}
