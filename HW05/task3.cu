#include <cstdio>
#include <cstdlib>
#include <random>
#include "vscale.cuh"
#include <chrono>

using std::chrono::high_resolution_clock;
using std::chrono::duration;

int main(int argc, char *argv[]) {

    int N = std::stoi(argv[1]);

    int some_seed = 759;
    std::mt19937 generator(some_seed);

    std::uniform_real_distribution<float> adist(-10., 10.);
    std::uniform_real_distribution<float> bdist(0., 1.);

    float hA[N], hB[N];
    
    for (int i = 0; i < N; i++) {
        hA[i] = adist(generator);
        hB[i] = bdist(generator);
    }
    
    
    
    float *dA, *dB;
    hipMalloc((void**)&dA, N * sizeof(float));
    hipMalloc((void**)&dB, N * sizeof(float));

    hipMemcpy(dA, hA, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = std::stoi(argv[2]);;
    if (N < threadsPerBlock) {
        threadsPerBlock = N;
    }
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    high_resolution_clock::time_point start;
    high_resolution_clock::time_point end;
    duration<double, std::milli> duration_sec;

    start = high_resolution_clock::now();
    vscale<<<blocks, threadsPerBlock>>>(dA, dB, N);

    hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hB, dB, N * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    end = high_resolution_clock::now();

    duration_sec = std::chrono::duration_cast<duration<double, std::milli>>(end - start);

    printf("%f\n",duration_sec.count());
    printf("%f\n",hB[0]);
    printf("%f\n",hB[N - 1]);

    

    hipFree(dA);
    hipFree(dB);

    return 0;
}
