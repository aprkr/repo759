#include <cstdio>
#include <hip/hip_runtime.h>

// Kernel function to compute factorial of integers 1 to 8
__global__ void computeFactorial() {
    int index = threadIdx.x + 1; // Thread index: 1 to 8
    unsigned int fact = 1;

    // Compute factorial of index
    for (int i = 1; i <= index; i++) {
        fact *= i;
    }

    // Print result in the form: a!=b
    printf("%d!=%u\n", index, fact);
}

int main() {
    // Launch the kernel with 1 block and 8 threads
    computeFactorial<<<1, 8>>>();

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    return 0;
}
