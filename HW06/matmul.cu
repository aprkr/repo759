#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n) {
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    size_t col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (size_t k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block) {
    int blocks = (n + threads_per_block - 1) / threads_per_block;

    float *d_A, *d_B, *d_C;
    size_t num_bytes = n * n * sizeof(float);
    hipMalloc(&d_A, num_bytes);
    hipMalloc(&d_B, num_bytes);
    hipMalloc(&d_C, num_bytes);

    hipMemcpy(d_A, A, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, num_bytes, hipMemcpyHostToDevice);

    matmul_kernel<<<blocks, threads_per_block>>>(d_A, d_B, d_C, n);
    
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, num_bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
