// Generated with ChatGPT
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <random>
#include <hip/hip_runtime.h>
#include "matmul.cuh"

void fill_random(float* mat, size_t n) {
    int some_seed = 759;
    std::mt19937 generator(some_seed);

    std::uniform_real_distribution<float> adist(-1., 1.);

    for (size_t i = 0; i < n * n; ++i) {
        mat[i] = adist(generator);
    }
}

int main(int argc, char** argv) {
    size_t n = std::stoul(argv[1]);
    unsigned int threads_per_block = std::stoul(argv[2]);

    float* A = new float[n * n];
    float* B = new float[n * n];
    float* C = new float[n * n];

    fill_random(A, n);
    fill_random(B, n);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matmul(A, B, C, n, threads_per_block);

    hipEventRecord(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << C[n * n - 1] << std::endl;

    std::cout << milliseconds << std::endl;
}
