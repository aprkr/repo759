// Generated with ChatGPT
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include "matmul.cuh"

void fill_random(float* mat, size_t n) {
    for (size_t i = 0; i < n * n; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f;
    }
}

int main(int argc, char** argv) {
    size_t n = std::stoul(argv[1]);
    unsigned int threads_per_block = std::stoul(argv[2]);

    float* A = new float[n * n];
    float* B = new float[n * n];
    float* C = new float[n * n];

    srand(static_cast<unsigned int>(time(0)));
    fill_random(A, n);
    fill_random(B, n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_B, n * n * sizeof(float));
    hipMalloc(&d_C, n * n * sizeof(float));

    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matmul(d_A, d_B, d_C, n, threads_per_block);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << C[n * n - 1] << std::endl;

    std::cout << milliseconds << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] A;
    delete[] B;
    delete[] C;
}
