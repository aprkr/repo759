// Generated with ChatGPT
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include "matmul.cuh"

void fill_random(float* mat, size_t n) {
    for (size_t i = 0; i < n * n; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f;
    }
}

int main(int argc, char** argv) {
    size_t n = std::stoul(argv[1]);
    unsigned int threads_per_block = std::stoul(argv[2]);

    float* A = new float[n * n];
    float* B = new float[n * n];
    float* C = new float[n * n];

    srand(static_cast<unsigned int>(time(0)));
    fill_random(A, n);
    fill_random(B, n);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matmul(A, B, C, n, threads_per_block);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << C[n * n - 1] << std::endl;

    std::cout << milliseconds << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] A;
    delete[] B;
    delete[] C;
}
