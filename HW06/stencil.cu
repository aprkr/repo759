#include "hip/hip_runtime.h"

#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel function for performing the stencil operation
__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
    // Shared memory: first part will store the mask, second part will store the image window
    extern __shared__ float shared_mem[];

    // Shared memory pointers
    float* shared_mask = shared_mem;          // Shared memory for the mask
    float* shared_image = shared_mem + (2 * R + 1); // Shared memory for the image window

    // Thread index within the block
    unsigned int thread_idx = threadIdx.x;

    // Load the mask into shared memory (all threads load the full mask)
    if (thread_idx < (2 * R + 1)) {
        shared_mask[thread_idx] = mask[thread_idx];
    }

    // Global index of the element to be processed
    unsigned int global_idx = blockIdx.x * blockDim.x + thread_idx - R;

    // Load image elements into shared memory
    if (global_idx >= 0 && global_idx < n) {
        shared_image[thread_idx] = image[global_idx];
    } else {
        shared_image[thread_idx] = 1.0f; // Boundary handling (image[i] = 1 for out-of-bounds)
    }

    __syncthreads(); // Synchronize to make sure image and mask are loaded before computation

    // Compute the output value if the global index is within bounds
    if (global_idx >= 0 && global_idx < n) {
        float result  0.0f;
        // Perform convolution: sum(mask[j] * image[i+j]) for j = -R to R
        for (int j = -R; j <= R; ++j) {
            int image_idx = thread_idx + j;
            result += shared_image[image_idx] * shared_mask[j + R];
        }
        output[global_idx] = result;
    }
}

// Host function that launches the stencil kernel
void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block) {
    // Allocate device memory for image, mask, and output arrays
    float *d_image, *d_mask, *d_output;
    hipMalloc(&d_image, n * sizeof(float));
    hipMalloc(&d_mask, (2 * R + 1) * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_image, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid size: the number of blocks needed
    unsigned int blocks = (n + threads_per_block - 1) / threads_per_block;

    // Launch the stencil kernel
    size_t shared_mem_size = (2 * R + 1 + threads_per_block) * sizeof(float);  // Shared memory size for both image window and mask
    stencil_kernel<<<blocks, threads_per_block, shared_mem_size>>>(d_image, d_mask, d_output, n, R);

    // Check for any CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel failed: " << hipGetErrorString(err) << std::endl;
    }

    // Copy result from device to host
    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
}

