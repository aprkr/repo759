#include "hip/hip_runtime.h"

#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel function for performing the stencil operation
__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
    // Shared memory to store the mask and the image window for the block
    extern __shared__ float shared_mem[];

    // Mask is stored in the first part of shared memory
    float* shared_mask = shared_mem;
    // Image window for the current block is stored after the mask
    float* shared_image = shared_mem + (2 * R + 1);

    // Thread index within the block
    unsigned int thread_idx = threadIdx.x;

    // Load the mask into shared memory
    if (thread_idx < (2 * R + 1)) {
        shared_mask[thread_idx] = mask[thread_idx];
    }

    // Load the image elements needed for convolution into shared memory
    unsigned int global_idx = blockIdx.x * blockDim.x + thread_idx - R;
    if (global_idx >= 0 && global_idx < n) {
        shared_image[thread_idx] = image[global_idx];
    } else {
        shared_image[thread_idx] = 1.0f; // Outside the image boundary, set image value to 1
    }

    __syncthreads();  // Synchronize threads to ensure shared memory is loaded

    // Each thread computes one element of the output array
    if (global_idx >= 0 && global_idx < n) {
        float result = 0.0f;
        for (int j = -R; j <= R; ++j) {
            result += shared_image[thread_idx + j] * shared_mask[j + R];
        }
        output[global_idx] = result;
    }
}

// Host function that launches the stencil kernel
void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block) {
    // Allocate device memory for image, mask, and output arrays
    float *d_image, *d_mask, *d_output;
    hipMalloc(&d_image, n * sizeof(float));
    hipMalloc(&d_mask, (2 * R + 1) * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_image, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid size: the number of blocks needed
    unsigned int blocks = (n + threads_per_block - 1) / threads_per_block;

    // Launch the stencil kernel
    size_t shared_mem_size = (2 * R + 1 + threads_per_block) * sizeof(float);  // Shared memory size for both image window and mask
    stencil_kernel<<<blocks, threads_per_block, shared_mem_size>>>(d_image, d_mask, d_output, n, R);

    // Check for any CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel failed: " << hipGetErrorString(err) << std::endl;
    }

    // Copy result from device to host
    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
}

